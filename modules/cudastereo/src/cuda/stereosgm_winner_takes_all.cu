#include "hip/hip_runtime.h"
/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include "stereosgm_winner_takes_all.hpp"
#include "stereosgm_utility.hpp"
#include "opencv2/core/cuda_stream_accessor.hpp"
#include "opencv2/calib3d.hpp"
#include "opencv2/cudev/warp/reduce.hpp"
#include "opencv2/cudev/warp/shuffle.hpp"

namespace cv { namespace cuda { namespace device
{
namespace stereosgm
{
namespace {
static constexpr unsigned int NUM_PATHS = 8u;

static constexpr unsigned int WARP_SIZE = 32u;
static constexpr unsigned int WARPS_PER_BLOCK = 8u;
static constexpr unsigned int BLOCK_SIZE = WARPS_PER_BLOCK * WARP_SIZE;

struct OpAnd
{
    __device__ __forceinline__ bool operator()(bool x, bool y) const
    {
        return x && y;
    }
};
static constexpr OpAnd OP_AND{};

struct OpMin
{
    __device__ __forceinline__ uint32_t operator()(uint32_t x, uint32_t y) const
    {
        return ::min(x, y);
    }
};
static constexpr OpMin OP_MIN{};

__device__ inline uint32_t pack_cost_index(uint32_t cost, uint32_t index){
    union {
        uint32_t uint32;
        ushort2 uint16x2;
    } u;
    u.uint16x2.x = static_cast<uint16_t>(index);
    u.uint16x2.y = static_cast<uint16_t>(cost);
    return u.uint32;
}

__device__ uint32_t unpack_cost(uint32_t packed){
    return packed >> 16;
}

__device__ int unpack_index(uint32_t packed){
    return packed & 0xffffu;
}

using ComputeDisparity = uint32_t(*)(uint32_t, uint32_t, uint16_t*);

__device__ inline uint32_t compute_disparity_normal(uint32_t disp, uint32_t cost = 0, uint16_t* smem = nullptr)
{
    return disp;
}

template <size_t MAX_DISPARITY>
__device__ inline uint32_t compute_disparity_subpixel(uint32_t disp, uint32_t cost, uint16_t* smem)
{
    uint32_t subp = disp;
    subp <<= StereoMatcher::DISP_SHIFT;
    if (disp > 0 && disp < MAX_DISPARITY - 1)
    {
        const int left = smem[disp - 1];
        const int right = smem[disp + 1];
        const int numer = left - right;
        const int denom = left - 2 * cost + right;
        subp += ((numer << StereoMatcher::DISP_SHIFT) + denom) / (2 * denom);
    }
    return subp;
}


template <unsigned int MAX_DISPARITY, ComputeDisparity compute_disparity = compute_disparity_normal>
__global__ void winner_takes_all_kernel(
    const PtrStep<uint8_t> _src,
    PtrStep<int16_t> _left_dest,
    PtrStep<int16_t> _right_dest,
    int width,
    int height,
    float uniqueness)
{
    static const unsigned int ACCUMULATION_PER_THREAD = 16u;
    static const unsigned int REDUCTION_PER_THREAD = MAX_DISPARITY / WARP_SIZE;
    static const unsigned int ACCUMULATION_INTERVAL = ACCUMULATION_PER_THREAD / REDUCTION_PER_THREAD;
    static const unsigned int UNROLL_DEPTH =
        (REDUCTION_PER_THREAD > ACCUMULATION_INTERVAL)
            ? REDUCTION_PER_THREAD
            : ACCUMULATION_INTERVAL;

    const unsigned int cost_step = MAX_DISPARITY * width * height;
    const unsigned int warp_id = threadIdx.x / WARP_SIZE;
    const unsigned int lane_id = threadIdx.x % WARP_SIZE;

    const unsigned int y = blockIdx.x * WARPS_PER_BLOCK + warp_id;
    const PtrStep<uint8_t> src{(uint8_t*)&_src(0, y * MAX_DISPARITY * width), height * width * MAX_DISPARITY * NUM_PATHS};
    PtrStep<int16_t> left_dest{_left_dest.ptr(y), _left_dest.step};
    PtrStep<int16_t> right_dest{_right_dest.ptr(y), _right_dest.step};

    if(y >= height){
        return;
    }

    __shared__ uint16_t smem_cost_sum[WARPS_PER_BLOCK][ACCUMULATION_INTERVAL][MAX_DISPARITY];

    uint32_t right_best[REDUCTION_PER_THREAD];
    for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
        right_best[i] = 0xffffffffu;
    }

    for(unsigned int x0 = 0; x0 < width; x0 += UNROLL_DEPTH){
#pragma unroll
        for(unsigned int x1 = 0; x1 < UNROLL_DEPTH; ++x1){
            if(x1 % ACCUMULATION_INTERVAL == 0){
                const unsigned int k = lane_id * ACCUMULATION_PER_THREAD;
                const unsigned int k_hi = k / MAX_DISPARITY;
                const unsigned int k_lo = k % MAX_DISPARITY;
                const unsigned int x = x0 + x1 + k_hi;
                if(x < width){
                    const unsigned int offset = x * MAX_DISPARITY + k_lo;
                    uint32_t sum[ACCUMULATION_PER_THREAD];
                    for(unsigned int i = 0; i < ACCUMULATION_PER_THREAD; ++i){
                        sum[i] = 0;
                    }
                    for(unsigned int p = 0; p < NUM_PATHS; ++p){
                        uint32_t load_buffer[ACCUMULATION_PER_THREAD];
                        load_uint8_vector<ACCUMULATION_PER_THREAD>(
                            load_buffer, &src(0, p * cost_step + offset));
                        for(unsigned int i = 0; i < ACCUMULATION_PER_THREAD; ++i){
                            sum[i] += load_buffer[i];
                        }
                    }
                    store_uint16_vector<ACCUMULATION_PER_THREAD>(
                        &smem_cost_sum[warp_id][k_hi][k_lo], sum);
                }
#if CUDA_VERSION >= 9000
                __syncwarp();
#else
                __threadfence_block();
#endif
            }
            const unsigned int x = x0 + x1;
            if(x < width){
                // Load sum of costs
                const unsigned int smem_x = x1 % ACCUMULATION_INTERVAL;
                const unsigned int k0 = lane_id * REDUCTION_PER_THREAD;
                uint32_t local_cost_sum[REDUCTION_PER_THREAD];
                load_uint16_vector<REDUCTION_PER_THREAD>(
                    local_cost_sum, &smem_cost_sum[warp_id][smem_x][k0]);
                // Pack sum of costs and dispairty
                uint32_t local_packed_cost[REDUCTION_PER_THREAD];
                for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
                    local_packed_cost[i] = pack_cost_index(local_cost_sum[i], k0 + i);
                }
                // Update left
                uint32_t best = 0xffffffffu;
                for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
                    best = ::min(best, local_packed_cost[i]);
                }
            #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 300)
                cv::cudev::warpReduce<uint32_t, OpMin>(nullptr, best, 0, OP_MIN);
                best = cv::cudev::shfl(best, 0);
            #else
                // TODO
            #endif
                // Update right
#pragma unroll
                for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
                    const unsigned int k = lane_id * REDUCTION_PER_THREAD + i;
                    const int p = static_cast<int>(((x - k) & ~(MAX_DISPARITY - 1)) + k);
                    const unsigned int d = static_cast<unsigned int>(x - p);
#if CUDA_VERSION >= 9000
                    const uint32_t recv = __shfl_sync(0xffffffffu,
                        local_packed_cost[(REDUCTION_PER_THREAD - i + x1) % REDUCTION_PER_THREAD],
                        d / REDUCTION_PER_THREAD,
                        WARP_SIZE);
#else
                    const uint32_t recv = __shfl(
                        local_packed_cost[(REDUCTION_PER_THREAD - i + x1) % REDUCTION_PER_THREAD],
                        d / REDUCTION_PER_THREAD,
                        WARP_SIZE);
#endif
                    right_best[i] = ::min(right_best[i], recv);
                    if(d == MAX_DISPARITY - 1){
                        if(0 <= p){
                            right_dest(0, p) = compute_disparity_normal(unpack_index(right_best[i]));
                        }
                        right_best[i] = 0xffffffffu;
                    }
                }
                // Resume updating left to avoid execution dependency
                const uint32_t bestCost = unpack_cost(best);
                const int bestDisp = unpack_index(best);
                bool uniq = true;
                for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
                    const uint32_t x = local_packed_cost[i];
                    const bool uniq1 = unpack_cost(x) * uniqueness >= bestCost;
                    const bool uniq2 = ::abs(unpack_index(x) - bestDisp) <= 1;
                    uniq &= uniq1 || uniq2;
                }
                cv::cudev::warpReduce<bool, OpAnd>(nullptr, uniq, 0, OP_AND);
                if(lane_id == 0){
                    left_dest(0, x) = uniq ? compute_disparity(bestDisp, bestCost, smem_cost_sum[warp_id][smem_x]) : 0;
                }
            }
        }
    }
    for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
        const unsigned int k = lane_id * REDUCTION_PER_THREAD + i;
        const int p = static_cast<int>(((width - k) & ~(MAX_DISPARITY - 1)) + k);
        if(p < width){
            right_dest(0, p) = compute_disparity_normal(unpack_index(right_best[i]));
        }
    }
}
}

template <size_t MAX_DISPARITY>
void winnerTakesAll(const GpuMat& src, GpuMat& left, GpuMat& right, float uniqueness, bool subpixel, cv::cuda::Stream& _stream)
{
    cv::Size size = left.size();
    CV_Assert(src.rows == 1 && src.cols == size.width * size.height * MAX_DISPARITY * NUM_PATHS);
    CV_Assert(size == right.size());
    CV_Assert(left.type() == right.type());
    CV_Assert(src.type() == CV_8UC1);
    const int gdim =
        (size.height + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    const int bdim = BLOCK_SIZE;
    hipStream_t stream = cv::cuda::StreamAccessor::getStream(_stream);
    if (subpixel) {
        winner_takes_all_kernel<MAX_DISPARITY, compute_disparity_subpixel<MAX_DISPARITY>><<<gdim, bdim, 0, stream>>>(
            src, left, right, size.width, size.height, uniqueness);
    } else {
        winner_takes_all_kernel<MAX_DISPARITY, compute_disparity_normal><<<gdim, bdim, 0, stream>>>(
            src, left, right, size.width, size.height, uniqueness);
    }
}
template CV_EXPORTS_W void winnerTakesAll< 64>(const GpuMat&, GpuMat&, GpuMat&, float, bool, cv::cuda::Stream&);
template CV_EXPORTS_W void winnerTakesAll<128>(const GpuMat&, GpuMat&, GpuMat&, float, bool, cv::cuda::Stream&);
}
}}}
